
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){
    printf("Hello CUDA World !!\n");
}

int main() {
    hello<<< 2, 4 >>>();
    hipDeviceSynchronize();
    return 0;
}
