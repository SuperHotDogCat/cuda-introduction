#include <stdio.h>
#include <hip/hip_runtime.h>

/*
Naming rule: function: Upper Camel, Variable: snake 
*/

__device__ void gpuAdd(int *number){
    *number += 1;
}

__global__ void callGpu(int *number){
    gpuAdd(number);
}

int main(){

    //device set up
    int device_id = 0;
    hipSetDevice(device_id);
    //allocate memory on cpu
    int *g = (int*)malloc(sizeof(int));
    *g = 0;
    // allocate memory on gpu
    int *d_g = 0;
    hipMalloc((void**) &d_g,sizeof(int));
    // memcpy host -> device
    hipMemcpy(d_g, g, sizeof(int), hipMemcpyHostToDevice); //これがUpperCaseじゃないのまじで納得行ってない
    // execute
    callGpu<<<1, 1>>>(d_g);
    hipDeviceSynchronize(); // Wait until GPU processing finishs.
    hipMemcpy(g, d_g, sizeof(int), hipMemcpyDeviceToHost); 
    // free
    hipFree(d_g);
    // display the answer
    printf("ans: %d \n", *g);
    return 0;
}